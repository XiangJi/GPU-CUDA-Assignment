#include <stdio.h>  // C standard I/O header
#include <sys/time.h> // system time
#include <hip/hip_runtime.h> //Defines the public host functions and types for the CUDA API
#include <cfloat> //C float.h
#include <math.h>

// The number of threads per blocks in the kernel
// (if we define it here, then we can use its value in the kernel,
//  for example to statically declare an array in shared memory)
const int threads_per_block = 256;


// Forward function declarations
float GPU_vector_max(float *A, int N, int kernel_code);
float CPU_vector_max(float *A, int N);
float *get_random_vector(int N);
float *get_increasing_vector(int N);
long long start_timer();
long long stop_timer(long long start_time, char *name);
void die(char *message);
void checkError();

int main(int argc, char **argv) {

    //default kernel
    int kernel_code = 1;
    //argc = number of arguements, argv = arguement string
    //sample arguements  vectormax 256 -k 1 [0123]
    // Parse vector length and kernel options, arguement listener
    int N;
    if(argc == 2) {
        N = atoi(argv[1]); // user-specified value
    } else if (argc == 4 && !strcmp(argv[2], "-k")) {
        N = atoi(argv[1]); // user-specified value
        kernel_code = atoi(argv[3]); 
        printf("KERNEL_CODE %d\n", kernel_code); //decimal output
    } else {
        die("USAGE: ./vector_max <vector_length> -k <kernel_code>");//otherwise promote usage
    }

    // Seed the random generator (use a constant here for repeatable results)
    srand(10);//generate seed for rand function to get random number

    // Generate a random vector
    // You can use "get_increasing_vector()" for debugging, vector with order
    long long vector_start_time = start_timer(); // longlong 64 bits variable
    float *vec = get_random_vector(N); // get random vector of N, rand() inside

    //float *vec = get_increasing_vector(N); just for debugging
    stop_timer(vector_start_time, "Vector generation");
	
    // Compute the max on the GPU
    long long GPU_start_time = start_timer();
    float result_GPU = GPU_vector_max(vec, N, kernel_code);
    long long GPU_time = stop_timer(GPU_start_time, "\t            Total");// t = tab
	
    // Compute the max on the CPU
    long long CPU_start_time = start_timer();
    float result_CPU = CPU_vector_max(vec, N);
    long long CPU_time = stop_timer(CPU_start_time, "\nCPU");
    
    // Free vector, release the memory for vec, opposite to malloc
    free(vec);

    // Compute the speedup or slowdown
    if (GPU_time > CPU_time) printf("\nCPU outperformed GPU by %.2fx\n", (float) GPU_time / (float) CPU_time);
    else                     printf("\nGPU outperformed CPU by %.2fx\n", (float) CPU_time / (float) GPU_time);
	
    // Check the correctness of the GPU result, CPU must be correct
    int wrong = result_CPU != result_GPU;
	
    // Report the correctness results
    if(wrong) printf("GPU output %f did not match CPU output %f\n", result_GPU, result_CPU);
        
}


// A GPU kernel that computes the maximum value of a vector, some as loop for computing max
// (each lead thread (threadIdx.x == 0) computes a single value, parallel kernel
__global__ void vector_max_kernel1(float *in, float *out, int N) {

    // Determine the "flattened" block id and thread id, dim3 and unit3, still number but unassigned
    int block_id = blockIdx.x + gridDim.x * blockIdx.y;
    int thread_id = blockDim.x * block_id + threadIdx.x;

    // A single "lead" thread in each block finds the maximum value over a range of size threads_per_block, only use one thread in one block
    float max = 0.0;
    if (threadIdx.x == 0) {

        //calculate out of bounds guard, vague, actually the remained threads in on block
        //our block size will be 256, but our vector may not be a multiple of 256!
        int end = threads_per_block;
        if(thread_id + threads_per_block > N)
            end = N - thread_id;

        //grab the lead thread's value, in[] is the floast of the element in vector
        max = in[thread_id];

        //grab values from all other threads' locations, obtain max in every block
        for(int i = 1; i < end; i++) {
                
            //if larger, replace
            if(max < in[thread_id + i])
                max = in[thread_id + i];
        }

        out[block_id] = max; // store every the the biggest value in all blocks

    }
}

__global__ void vector_max_kernel2(float *in, float *out, int N) {
	//allocate a shared memory in block
	__shared__ float sharedmem[threads_per_block];
    // Determine the "flattened" block id and thread id, dim3 and unit3, still number but unassigned
    int block_id = blockIdx.x + gridDim.x * blockIdx.y;
    int thread_id = blockDim.x * block_id + threadIdx.x;
	
	//copy vector to each shared memory of each block;
	sharedmem[threadIdx.x] = in[thread_id];
	__syncthreads();

    // A single "lead" thread in each block finds the maximum value over a range of size threads_per_block, only use one thread in one block
    float max = 0.0;
    if (threadIdx.x == 0) {

        //calculate out of bounds guard, vague, actually the remained threads in on block
        //our block size will be 256, but our vector may not be a multiple of 256!
        int end = threads_per_block;
        if(thread_id + threads_per_block > N)
            end = N - thread_id;

        //grab the lead thread's value, in[] is the floast of the element in vector
        max = sharedmem[threadIdx.x];

        //grab values from all other threads' locations, obtain max in every block
        for(int i = 1; i < end; i++) {
                
            //always
            if(max < sharedmem[threadIdx.x + i])
                max = sharedmem[threadIdx.x + i];
        }

        out[block_id] = max; // store every the the biggest value in all blocks

    }
}


__global__ void vector_max_kernel3(float *in, float *out, int N) {
	//allocate a shared memory in block
	__shared__ float sharedmem[threads_per_block];
	__shared__ int end;
    // Determine the "flattened" block id and thread id, dim3 and unit3, still number but unassigned
    int block_id = blockIdx.x + gridDim.x * blockIdx.y;
    int thread_id = blockDim.x * block_id + threadIdx.x;
	
	sharedmem[threadIdx.x] = 0;
	//copy vector to each shared memory of each block;
	sharedmem[threadIdx.x] = in[thread_id];
	__syncthreads();

    // A single "lead" thread in each block finds the maximum value over a range of size threads_per_block, only use one thread in one block
    if (threadIdx.x == 0) {
        //calculate out of bounds guard, vague, actually the remained threads in on block
        //our block size will be 256, but our vector may not be a multiple of 256!
        end = threads_per_block;
        if(thread_id + threads_per_block > N)
            end = N - thread_id;
		if(end % 2 != 0)
			end += 1;
	}
	__syncthreads();
	
	 //grab values from all other threads' locations, obtain max in every block
    for(int i = 1; i < end; i = i * 2) {
		if (threadIdx.x % (2 * i) == 0) {
			//alway put bigger one in the lower position, step times 2 every iteration
            if(sharedmem[threadIdx.x] < sharedmem[threadIdx.x + i])
                sharedmem[threadIdx.x] = sharedmem[threadIdx.x + i];
		}
		__syncthreads();
    }
		
	if (threadIdx.x == 0)
		out[block_id] = sharedmem[0]; // put the max one in [0] for outblock
	
}
//group useful threads
__global__ void vector_max_kernel4(float *in, float *out, int N) {
	//allocate a shared memory in block
	__shared__ float sharedmem[threads_per_block];
	__shared__ int end;
    // Determine the "flattened" block id and thread id, dim3 and unit3, still number but unassigned
    int block_id = blockIdx.x + gridDim.x * blockIdx.y;
    int thread_id = blockDim.x * block_id + threadIdx.x;
	
	sharedmem[threadIdx.x] = 0;
	//copy vector to each shared memory of each block;
	sharedmem[threadIdx.x] = in[thread_id];
	__syncthreads();

    // A single "lead" thread in each block finds the maximum value over a range of size threads_per_block, only use one thread in one block
    if (threadIdx.x == 0) {
        //calculate out of bounds guard, vague, actually the remained threads in on block
        //our block size will be 256, but our vector may not be a multiple of 256!
        end = threads_per_block;
        if(thread_id + threads_per_block > N)
            end = N - thread_id;
		
		end = (int)powf(2, ceilf(log2f((float)end)));
	}
	__syncthreads();
	
	int tblock = end; 
	while (tblock > 1){
		if (threadIdx.x < tblock / 2) {
			if(sharedmem[threadIdx.x] < sharedmem[threadIdx.x + tblock/2])
				sharedmem[threadIdx.x] = sharedmem[threadIdx.x + tblock/2];
		}
		tblock /= 2;
		__syncthreads();
    }
		
	if (threadIdx.x == 0)
		out[block_id] = sharedmem[0]; // put the max one in [0] for outblock
	
}



// Returns the maximum value within a vector of length N, use GPU method
float GPU_vector_max(float *in_CPU, int N, int kernel_code) {

    int vector_size = N * sizeof(float);//size of float

    // Allocate CPU memory for the result, give OUT_CPU space
    float *out_CPU = (float *) malloc(vector_size);
    if (out_CPU == NULL) die("Error allocating CPU memory");

    // Allocate GPU memory for the inputs and the result
    long long memory_start_time = start_timer();

    float *in_GPU, *out_GPU;// the threads vector and the block vector
    if (hipMalloc((void **) &in_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
    if (hipMalloc((void **) &out_GPU, vector_size) != hipSuccess) die("Error allocating GPU memory");
    //cudaSuccess is a error variable which record the error
    //cudaPeekAtLastError() returns this variable. cudaGetLastError() returns this variable and resets it to cudaSuccess

    // Transfer the input vectors to GPU memory
    hipMemcpy(in_GPU, in_CPU, vector_size, hipMemcpyHostToDevice);// dst, src, size, kind
    hipDeviceSynchronize();  //synchronize just after the call, check for asynchronous errors, here only timing purpose
    stop_timer(memory_start_time, "\nGPU:\t  Transfer to GPU");// transfer time

    bool lastBlock = 0;
    while (!lastBlock) {
        // Determine the number of thread blocks in the x- and y-dimension
        int num_blocks = (int) ((float) (N + threads_per_block - 1) / (float) threads_per_block);
        int max_blocks_per_dimension = 65535;
        int num_blocks_y = (int) ((float) (num_blocks + max_blocks_per_dimension - 1) / (float) max_blocks_per_dimension);
        int num_blocks_x = (int) ((float) (num_blocks + num_blocks_y - 1) / (float) num_blocks_y);
        dim3 grid_size(num_blocks_x, num_blocks_y, 1);

        // Execute the kernel to compute the vector sum on the GPU
        long long kernel_start_time;
        kernel_start_time = start_timer();

        switch(kernel_code){
        case 1 :
            vector_max_kernel1 <<< grid_size , threads_per_block >>> (in_GPU, out_GPU, N);
            break;
        case 2 :
            vector_max_kernel2 <<< grid_size , threads_per_block >>> (in_GPU, out_GPU, N);
            break;
        case 3 :
            vector_max_kernel3 <<< grid_size , threads_per_block >>> (in_GPU, out_GPU, N);
            break;
        case 4 :
            //LAUNCH KERNEL FROM PROBLEM 4 HERE
            vector_max_kernel4 <<< grid_size , threads_per_block >>> (in_GPU, out_GPU, N);
            break;
        default :
            die("INVALID KERNEL CODE\n");
        }

        if (num_blocks > 1) {
            lastBlock = 0;
            N = num_blocks;
            hipMemcpy(in_GPU, out_GPU, vector_size, hipMemcpyDeviceToDevice);
            hipDeviceSynchronize();
        }
        else {
            lastBlock = 1;
        }

        hipDeviceSynchronize();  // this is only needed for timing purposes
        stop_timer(kernel_start_time, "\t Kernel execution");

        checkError();
    }
    



    // Transfer the result from the GPU to the CPU
    memory_start_time = start_timer();
    
    //copy C back from GPU device to CPU
    hipMemcpy(out_CPU, out_GPU, vector_size, hipMemcpyDeviceToHost);
    checkError();
    hipDeviceSynchronize();  // this is only needed for timing purposes
    stop_timer(memory_start_time, "\tTransfer from GPU");
    			    
    // Free the GPU memory
    hipFree(in_GPU);
    hipFree(out_GPU);

    float max = out_CPU[0];
    free(out_CPU);

    //return a single statistic, max in vector
    return max;
}


// Returns the maximum value within a vector of length N, just CPU simple MAX function
float CPU_vector_max(float *vec, int N) {	

    // find the max
    float max;
    max = vec[0];
    for (int i = 1; i < N; i++) {
        if(max < vec[i]) {
            max = vec[i];
        }
    }
	
    // Return a single statistic
    return max;
}


// Returns a randomized vector containing N elements, vector generator
float *get_random_vector(int N) {
    if (N < 1) die("Number of elements must be greater than zero");
	
    // Allocate memory for the vector, memory size N float, malloc
    float *V = (float *) malloc(N * sizeof(float));
    if (V == NULL) die("Error allocating CPU memory");
	
    // Populate the vector with random numbers
    for (int i = 0; i < N; i++) V[i] = (float) rand() / (float) rand();
	
    // Return the randomized vector, float *V
    return V;
}

float *get_increasing_vector(int N) {
    if (N < 1) die("Number of elements must be greater than zero");
	
    // Allocate memory for the vector
    float *V = (float *) malloc(N * sizeof(float));
    if (V == NULL) die("Error allocating CPU memory");
	
    // Populate the vector with random numbers, number fixed 1, 2, 3...
    for (int i = 0; i < N; i++) V[i] = (float) i;
	
    // Return the randomized vector
    return V;
}

//use it for debug kernel
void checkError() {
    // Check for kernel errors, based on cuda lib
    hipError_t error = hipGetLastError();
    if (error) {
        char message[256];
        sprintf(message, "CUDA error: %s", hipGetErrorString(error));
        die(message);
    }
}

// Returns the current time in microseconds, (us)
//int gettimeofday (struct timeval *tv, struct timezone *tz)
long long start_timer() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time, print and return the time interval
long long stop_timer(long long start_time, char *name) {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
    printf("%s: %.5f sec\n", name, ((float) (end_time - start_time)) / (1000 * 1000));
    return end_time - start_time;
}


// Prints the specified message and quits
void die(char *message) {
    printf("%s\n", message);
    exit(1); 
}
